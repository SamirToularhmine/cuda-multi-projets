#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>

#define PI 3.14159265


__global__ void rotate(unsigned char* rgb, unsigned char* g, std::size_t w, std::size_t h, float axeX, float axeY, double theta) {
	auto x1 = blockIdx.x * blockDim.x + threadIdx.x;
	auto y1= blockIdx.y * blockDim.y + threadIdx.y;
	float x0 = axeX; // axe de rotation de l'image en x
	float y0 = axeY; // axe de rotation de l'image en y


	theta = theta * PI / 180.0; // theta est en degré, on le passe donc en radian



	if (x1 < w && y1 < h) {
		int x2 = (int) (cos(theta) * (x1 - x0) + sin(theta) * (y1 - y0));   // nouvelle coordonnée
		int y2 = (int)(-sin(theta) * (x1 - x0) + cos(theta) * (y1 - y0));

		if (x2 >= 0 && x2 < w && y2 >= 0 && y2 < h) {
			g[3 * (y1 * w + x1)] = rgb[3 * (y2 * w + x2)];
			g[3 * (y1 * w + x1) + 1] = rgb[3 * (y2 * w + x2) + 1];
			g[3 * (y1 * w + x1) + 2] = rgb[3 * (y2 * w + x2) + 2];
		}

	}
}


int main()
{
	cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED);

	auto rgb = m_in.data; // rgb est l'image en rgb
	auto h = m_in.rows; // la hauteur
	auto w = m_in.cols; // la largeyr

	std::vector< unsigned char > s(w * h * 3); // s est l'image de sortie
	cv::Mat m_out(h, w, CV_8UC3, s.data()); // CV_8UC3 car image en RGB

	unsigned char* rgb_d = nullptr;
	unsigned char* g_d = nullptr;

	hipMalloc(&rgb_d, 3 * w * h); // *3 car rgb 
	hipMalloc(&g_d, 3 * w * h); //  *3 car rgb 

	hipMemcpy(rgb_d, rgb, 3 * w * h, hipMemcpyHostToDevice); //copie ilage rgb vers device

	dim3 block(32, 32); // a changer pour tester les configurations 32*4 // en général le premier cheiffre c'est 32
	dim3 grid((w - 1) / block.x + 1, (h - 1) / block.y + 1);


	rotate << < grid, block >> > (rgb_d, g_d, w, h, w, h, -180); //  w et h pour faire pivoter par rapport au coins en bas a droite et -180 pour mettre la photo à l'envers


	hipMemcpy(s.data(), g_d, w * h * 3, hipMemcpyDeviceToHost); //copie ilage rgb vers device

	//Copie image gris vers host

	cv::imwrite("out.jpg", m_out);

	hipFree(rgb_d);
	hipFree(g_d);

	return 0;
}